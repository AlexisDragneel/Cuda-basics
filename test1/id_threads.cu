#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <time.h>

__global__ void print_threadIds() {
	printf("threadIdx.x: %d, threadidx.y: %d, threadIdx.z: %d \n", threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void print_details() {
	printf("blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, blockDim.x: %d, blockDim.y: %d, blockDim.y: %d, gridDim.x: %d, gridDim.y: %d, gridDim.z: %d\n",
		blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

__global__ void print_3d_threads() {
	printf("threadIdx.x: %d, threadidx.y: %d, threadIdx.z: %d, gridDim.x: %d, gridDim.y: %d, gridDim.z: %d \n", 
		threadIdx.x, threadIdx.y, threadIdx.z, gridDim.x, gridDim.y, gridDim.z);

}

__global__ void unique_idx_calc_threadIdx(int* input) {
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	printf("threadIdx: %d, value: %d \n", tid, input[tid]);
}

__global__ void calculate_unique_gid() {
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;
	// index = tid + offset
	printf("threadIdx.x: %d, blockIdx.X: %d, blocDim.x: %d, gid: %d  \n",
		tid, blockIdx.x, blockDim.x, gid);
}

__global__ void unique_gid_calculation_2d_in_1d_array_1d_block(int* input ) {
	printf("threadIdx.x: %d, threadIdx.y: %d \n", threadIdx.x, threadIdx.y);
	printf("blockIdx.x: %d, blocIdx.y: %d \n", blockIdx.x, blockIdx.y);
	printf("blockDim.x: %d, blocDim.y: %d \n", blockDim.x, blockDim.y);
	printf("gid1d: %d, \n", threadIdx.x + (blockIdx.x + blockDim.x));

	// index = row offset + block ofsset  + tid;
	// index = number of threads in one row * blockIdx.y + number of threads in thread block * blockIdx.x + threadIdx.x
	// number of threads in one row = gridDim.x * blockDim.x
	int tid = threadIdx.x;
	int rowOffset = blockIdx.y * (gridDim.x * blockDim.x);
	int blockOffset = blockIdx.x * blockDim.x;
	int gid = tid + rowOffset + blockOffset;
	printf("gid: %d \n", gid);
	printf("value: %d \n", input[gid]);
}

__global__ void unique_gid_calculation_2d_in_1d_array_2d_block(int* input) {

	// tid = threadIdx.y * blockDim.x + threadIdx.x
	// block_offset = number of thread in a block (blockDim.x * blockDim.y) * blockIdx.x
	// row_offset = number of Threads in a row (blockDim.x blockDim.y * gridDim.x)  * blockIdx.y;
	// index = tid + block_offset + row_offset
	int tid = threadIdx.x + (blockDim.x * threadIdx.y);
	
	int num_of_threads_in_a_block = blockDim.x * blockDim.y;
	int block_offset = blockIdx.x * num_of_threads_in_a_block;

	int num_of_threads_in_a_row = num_of_threads_in_a_block * gridDim.x;
	int row_offset = num_of_threads_in_a_row * blockIdx.y;
	
	int gid = tid + block_offset + row_offset;
	printf("tid: %d, gid: %d, value: %d \n", tid, gid, input[gid]);
}

__global__ void mem_trs_test (int* input){
	int gid = threadIdx.x + (blockIdx.x + threadIdx.x);
	printf("tid: %d, gid: %d, value: %d \n", threadIdx.x, gid, input[gid]);
}

__global__ void mem_trs_test2(int* input, int size) {
	int gid = threadIdx.x + (blockIdx.x + threadIdx.x);
	if (gid < size)
		printf("tid: %d, gid: %d, value: %d \n", threadIdx.x, gid, input[gid]);
}

int main() {
	/**int nx, ny, nz;
	nx = 8;
	ny = 8;
	nz = 8;

	dim3 block(2, 2, 2);
	dim3 grid(nx / block.x, ny / block.y, nz/block.z);

	print_3d_threads << < grid, block>>> ();
	hipDeviceSynchronize();

	hipDeviceReset(); **/

	/**int array_size = 8;
	int array_byte_size = sizeof(int) * array_size;
	printf("%d \n",array_byte_size);
	int h_data[] = { 23,9,4,53,65,12,1,33 };

	for (int i = 0; i < array_size; i++) {
		printf("%d, ", h_data[i]);
	}

	printf("\n \n");

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice); **/

	/**int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	printf("%d \n", array_byte_size);
	int h_data[] = { 23,9,4,53,65,12,1,33, 23,9,4,53,65,12,1,33 };

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(2, 2);
	dim3 grid(2, 2);

	unique_gid_calculation_2d_in_1d_array_2d_block << <grid, block >> > (d_data);**/

	int size = 150;
	int byte_size = size * sizeof(int);

	int* h_input;
	h_input = (int*)malloc(byte_size);

	time_t t;
	srand((unsigned)time(&t));
	for (int i = 0; i < size; i++) {
		h_input[i] = (int)(rand() & 0xff);
	}

	int* d_input;
	hipMalloc((void**)&d_input, byte_size);
	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);
	dim3 block(32);
	dim3 grid(5);

	mem_trs_test2 << < grid, block >>> (d_input, size);

	hipDeviceSynchronize();

	hipDeviceReset();



	return 0;
}